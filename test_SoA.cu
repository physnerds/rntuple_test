#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include "SoALayout.h"
#include "SoAView.h"

#include<Eigen/Dense>
#include "Memory.cuh"

//#include "CUDACore/cudaCheck.h"


  GENERATE_SOA_LAYOUT_AND_VIEW(SoAHostDeviceLayoutTemplate,
                               SoAHostDeviceViewTemplate,
                               
  SOA_COLUMN(double, x),
  SOA_COLUMN(double, y),
  SOA_COLUMN(double, z),
  SOA_EIGEN_COLUMN(Eigen::Vector3d,a),
  SOA_EIGEN_COLUMN(Eigen::Vector3d,b),
  SOA_EIGEN_COLUMN(Eigen::Vector3d,r),
  SOA_SCALAR(const char*, description),
  SOA_SCALAR(uint32_t, someNumber)
  );

using SoAHostDeviceLayout = SoAHostDeviceLayoutTemplate<>;

using SoAHostDeviceView = SoAHostDeviceViewTemplate<cms::soa::CacheLineSize::NvidiaGPU, cms::soa::AlignmentEnforcement::Enforced>;


GENERATE_SOA_LAYOUT_AND_VIEW(SoADeviceOnlyLayoutTemplate,
                             SoADeviceOnlyViewTemplate,
                             SOA_COLUMN(uint16_t, color),
                             SOA_COLUMN(double, value),
                             SOA_COLUMN(double*, py),
                             SOA_COLUMN(uint32_t, count),
                             SOA_COLUMN(uint32_t, anotherCount))

using SoADeviceOnlyLayout = SoADeviceOnlyLayoutTemplate<>;
using SoADeviceOnlyView =
    SoADeviceOnlyViewTemplate<cms::soa::CacheLineSize::NvidiaGPU, cms::soa::AlignmentEnforcement::Enforced>;


// A 1 to 1 view of the store (except for unsupported types).
GENERATE_SOA_VIEW(SoAFullDeviceViewTemplate,
            SOA_VIEW_LAYOUT_LIST(SOA_VIEW_LAYOUT(SoAHostDeviceLayout, soaHD),
           SOA_VIEW_LAYOUT(SoADeviceOnlyLayout, soaDO)),
           SOA_VIEW_LAYOUT_LIST(SOA_VIEW_VALUE(soaHD, x),
           SOA_VIEW_VALUE(soaHD, y),
           SOA_VIEW_VALUE(soaHD, z),
           SOA_VIEW_VALUE(soaDO, color),
           SOA_VIEW_VALUE(soaDO, value),
           SOA_VIEW_VALUE(soaDO, py),
           SOA_VIEW_VALUE(soaDO, count),
           SOA_VIEW_VALUE(soaDO, anotherCount),
           SOA_VIEW_VALUE(soaHD, description),
           SOA_VIEW_VALUE(soaHD, someNumber)))

using SoAFullDeviceView =
    SoAFullDeviceViewTemplate<cms::soa::CacheLineSize::NvidiaGPU, cms::soa::AlignmentEnforcement::Enforced>;

__global__ void crossProduct(SoAHostDeviceView soa, const unsigned int numElements){
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if(i>=numElements);
  auto si = soa[i];
  si.r() = si.a().cross(si.b());

}

// Device-only producer kernel
__global__ void producerKernel(SoAFullDeviceView soa, const unsigned int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i>=numElements) return;
  auto si = soa[i];
  si.color() &= 0x55 << i % (sizeof(si.color()) - sizeof(char));
  si.value() = sqrt(si.x() * si.x() + si.y() * si.y() + si.z() * si.z());
}

// Device-only consumer with result in host-device area
__global__ void consumerKernel(SoAFullDeviceView soa, const unsigned int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i>=numElements) return;
  auto si = soa[i];
  si.x() = si.color() * si.value();
}



int main(int argc, char* argv[]){
  
  hipStream_t stream;
  CUDA_EXP_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
                 
  constexpr unsigned int numelements = 65537;
  
  size_t hostDeviceSize = SoAHostDeviceLayout::computeDataSize(numelements);
  
  std::cout<<"Size in Host "<<hostDeviceSize<<std::endl;
  
  //allocate buffers/ stores etc in host
  std::byte* h_buf = nullptr;
  
  CUDA_EXP_CHECK(hipHostMalloc(&h_buf, hostDeviceSize));
  SoAHostDeviceLayout h_soahdLayout(h_buf, numelements);
  SoAHostDeviceView h_soahd(h_soahdLayout);
  
  //allocate buffers, stores and views on device
  size_t deviceOnlySize = SoADeviceOnlyLayout::computeDataSize(numelements);
  std::cout<<"Size in Device "<<deviceOnlySize<<std::endl; 
  std::byte* d_buf = nullptr;
  CUDA_EXP_CHECK(hipHostMalloc(&d_buf,hostDeviceSize+deviceOnlySize));
    
  //create device layout and view..
  SoAHostDeviceLayout d_soahdLayout(d_buf,numelements);
  
  SoADeviceOnlyLayout d_soadoLayout(d_soahdLayout.soaMetadata().nextByte(),numelements);
  
  SoAHostDeviceView d_soahdView(d_soahdLayout);
  SoAFullDeviceView d_soaFullView(d_soahdLayout, d_soadoLayout);
 
    // Assert column alignments...do these for all SoA objects (view and layout)
  assert(0 == reinterpret_cast<uintptr_t>(h_soahd.soaMetadata().addressOf_x()) % decltype(h_soahd)::byteAlignment);
  
  std::memset(h_soahdLayout.soaMetadata().data(),0,hostDeviceSize);
  
  for(size_t i = 0;i<numelements;++i){
    auto si = h_soahd[i];
    si.x() = si.a()(0) = si.b()(2) = 1.0*i+1.0;
    si.y() = si.b()(1) = si.b()(1) = 2.0*i;
    si.z = si.a()(2) = si.b()(0) = 3.0*i-1.0;
  }
  auto& sn = h_soahd.someNumber();
  sn = numelements+2;
  
  CUDA_EXP_CHECK(hipMemcpyAsync(d_buf, h_buf,hostDeviceSize,hipMemcpyDefault,stream));
  
  crossProduct<<<(numelements+255)/256,256,0,stream>>>(d_soahdView,numelements);
  
  CUDA_EXP_CHECK(hipMemsetAsync(d_soadoLayout.soaMetadata().data(),0xFF,d_soadoLayout.soaMetadata().byteSize(),stream));
  
  producerKernel<<<(numelements+255)/256,256,0,stream>>>(d_soaFullView,numelements);
  
  consumerKernel<<<(numelements+255)/256,256,0,stream>>>(d_soaFullView,numelements);
  
  CUDA_EXP_CHECK(hipMemcpyAsync(h_buf,d_buf,hostDeviceSize,hipMemcpyDefault,stream));
  
 //uses the memory address to move data around.  
 CUDA_EXP_CHECK(hipStreamSynchronize(stream));
  
 for(size_t i =0;i<numelements;++i){
   auto si = h_soahd[i];
   assert(si.r()==si.a().cross(si.b()));
  // std::cout<<si.r()<<" "<<si.a().cross(si.b())<<std::endl;
 }
  return 1;
}