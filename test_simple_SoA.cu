#include "hip/hip_runtime.h"
#include "macros.h"
#include "val.h"
#include "Memory.cuh"
#include <iostream>

#define NTOT 1028*5

using namespace soa;


Generate_Arrays(soa_arrays,
                AddArray(int, xx, NTOT);
                AddArray(double, yy, NTOT);
                AddArray(uint32_t, zz, NTOT);
                );

Generate_Arrays(soa_simple,
                AddArray(myval<int>, xx,NTOT);
                AddArray(myval<double>,yy,NTOT);
                AddArray(myval<uint32_t>,zz,NTOT);
                );

//template wont work here...
__global__ void testVector(float a,soa_simple* C, soa_simple* A, soa_simple* B, int N=NTOT){
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  if(id<N){
  B->xx[id] = A->xx[id].val()*a+ C->xx[id].val();
  B->yy[id] = A->yy[id].val()*12.3+C->yy[id].val();
  B->zz[id] = A->zz[id].val()*2+C->zz[id].val();
  }
}


template<typename T>
__global__ void testvector(float a, T* C,T* A, T* B, int N=NTOT){
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  if(id<N){
  B->xx[id] = A->xx[id]*a+ C->xx[id];
  B->yy[id] = A->yy[id]*12.3+C->yy[id];
  B->zz[id] = A->zz[id]*2+C->zz[id];
  }
}


int test_soa(){
  int numThreads = NTOT;
  dim3 threadsPerBlock(1024,1,1);
  dim3 numberofBlocks((numThreads + threadsPerBlock.x-1)/threadsPerBlock.x,1,1);
    
  const int nentries = NTOT;
  soa_arrays Soa, Soaa;
  for(int i=0;i<nentries;i++){
    Soa.xx[i] = i;
    Soa.yy[i] = (i)*3.1415;
    Soa.zz[i] = i+8;
  }

  for(int i=0;i<nentries;i++){
    Soaa.xx[i] = i;
    Soaa.yy[i] = (i)*4.33;
    Soaa.zz[i] = i+81;
  }
  //create soa_simple objects for host and device..
  soa_arrays *d_Soa,*d_Soaa, *d_o_Soa, h_o_Soa;
  
  //allocate the memory....
  hipMalloc((void**)&d_Soa,sizeof(soa_arrays));
  hipMalloc((void**)&d_Soaa,sizeof(soa_arrays));
  
  hipMalloc((void**)&d_o_Soa,sizeof(soa_arrays));


  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipMemcpyAsync(d_Soa, &Soa, sizeof(soa_arrays),hipMemcpyHostToDevice);
  hipMemcpyAsync(d_Soaa, &Soaa, sizeof(soa_arrays),hipMemcpyHostToDevice);
 
  float a = 4.3f;
  hipEventRecord(start);
  testvector<<<threadsPerBlock,numberofBlocks>>>(a,d_Soaa,d_Soa, d_o_Soa);
  hipEventRecord(stop);  hipMemcpyAsync(&h_o_Soa,d_o_Soa,sizeof(soa_arrays),hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
//  for(int i=0;i<nentries;i++)std::cout<<h_o_Soa.xx[i]<<" "<<h_o_Soa.yy[i]<<" "<<h_o_Soa.zz[i]<<std::endl;
  auto var_size = sizeof(int)+sizeof(double)+sizeof(uint32_t);
  printf("Effective Bandwidth soa_arrays (GB/s): %f %f %d\n", NTOT*var_size*3/milliseconds/1e6, milliseconds,var_size);
  hipFree(d_o_Soa);
  hipFree(d_Soa);
  return 1;
}

int test_soa_simple(){ 
  int numThreads = NTOT;
  dim3 threadsPerBlock(1024,1,1);
  dim3 numberofBlocks((numThreads + threadsPerBlock.x-1)/threadsPerBlock.x,1,1);
    
  const int nentries = NTOT;
  soa_simple sa, saa;
  for(int i=0;i<nentries;i++){
    sa.xx[i] = i;
    sa.yy[i] = (i)*3.1415;
    sa.zz[i] = i+5;
  }
  
  for(int i=0;i<nentries;i++){
    saa.xx[i] = i;
    saa.yy[i] = (i)*4.33;
    saa.zz[i] = i+81;
  }

  //create soa_simple objects for host and device..
  soa_simple *d_Soa,*d_Soaa,*d_o_Soa, h_o_Soa;
  
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  //allocate the memory....
  hipMalloc((void**)&d_Soa,sizeof(soa_simple));
  hipMalloc((void**)&d_Soaa,sizeof(soa_simple));
  
  hipMalloc((void**)&d_o_Soa,sizeof(soa_simple));

  //now copy Soa content to d_Soa
  hipMemcpyAsync(d_Soa, &sa, sizeof(soa_simple),hipMemcpyHostToDevice);

  hipMemcpyAsync(d_Soaa, &saa, sizeof(soa_simple),hipMemcpyHostToDevice);
  
  float a = 4.3f;
  hipEventRecord(start);
  testVector<<<threadsPerBlock,numberofBlocks>>>(a,d_Soaa,d_Soa, d_o_Soa);
  hipEventRecord(stop);  
  hipEventSynchronize(stop);  hipMemcpyAsync(&h_o_Soa,d_o_Soa,sizeof(soa_simple),hipMemcpyDeviceToHost);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);  
 // for(int i=0;i<nentries;i++)std::cout<<i<<" "<<h_o_Soa.xx[i]<<" "<<h_o_Soa.yy[i]<<" "<<h_o_Soa.zz[i]<<std::endl;

  printf("Effective Bandwidth soa_simple (GB/s): %f %f %d\n", NTOT*128*3/milliseconds/1e6, milliseconds,128);  
  hipFree(d_o_Soa);
  hipFree(d_Soa);
  
  return 1;
}
int main(){
  
  test_soa_simple(); 
  test_soa();
  //1 means code exited with error code=1 (i.e. True)
  return EXIT_SUCCESS;
}